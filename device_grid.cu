#include <hip/hip_runtime.h>
extern "C" {
#include "grid.h"
}

extern "C" OTJ_Grid createDeviceGrid(OTJ_Grid host_grid) {
	OTJ_Grid device_grid = host_grid;
	hipMalloc(&device_grid.internal_storage, OTJ_Grid_Size(host_grid));
	return device_grid;
}

extern "C" OTJ_Grid createAndCopyDeviceGrid(OTJ_Grid host_grid) {
	OTJ_Grid device_grid = createDeviceGrid(host_grid);
	hipMemcpy(device_grid.internal_storage, host_grid.internal_storage, OTJ_Grid_Size(device_grid), hipMemcpyHostToDevice);
	return device_grid;
}

extern "C" void retrieveDeviceGrid(OTJ_Grid host_grid, OTJ_Grid device_grid) {
	hipMemcpy(host_grid.internal_storage, device_grid.internal_storage, OTJ_Grid_Size(device_grid), hipMemcpyDeviceToHost);
}
